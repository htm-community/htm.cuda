#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <algorithm>
#include <ctime>
#include <cstdlib>
#include <cmath>
#include <random>
#include <assert.h>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/fill.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>
#include <thrust/device_vector.h>

#include <thrust/random.h>
#include <thrust/transform.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/generate.h>

#include "SpatialPooler.cu"

#define checkError(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"CUDA error: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


using namespace std;

typedef unsigned int UInt;
typedef float Real;


UInt* generatePotentialPoolsUsingShuffle(UInt* potentialPools, const UInt SP_SIZE, const UInt IN_BLOCK_SIZE, const UInt MAX_CONNECTED) 
{
	
	vector<UInt> indeces(IN_BLOCK_SIZE);
	iota(indeces.begin(), indeces.end(), 0);

	// We could also do this on the device
	// thrust::host_vector<UInt> indeces(IN_BLOCK_SIZE);
	// thrust::sequence(input_indeces.begin(), input_indeces.end(), 0, 1);

	for(int i=0; i < SP_SIZE; i++) {
		random_shuffle(indeces.begin(), indeces.end());
		copy(indeces.begin(), indeces.begin()+MAX_CONNECTED, &potentialPools[i*MAX_CONNECTED]);
		// This may slightly improve performance, but slows down initialization
		sort(&potentialPools[i*MAX_CONNECTED], &potentialPools[(i+1)*MAX_CONNECTED]);
	}

	return potentialPools;
}

bool* generate01(bool* ar, size_t size, Real inDensity)
{
	for(int i=0; i < size; i++)
	{
		ar[i] = (Real)(rand()%100)/100 <= inDensity ? 1 : 0;
	}
	return ar;
}

struct prg : public thrust::unary_function<unsigned int,bool>
{
	Real IN_DENSITY;

	__host__ __device__
		prg(Real ind) : IN_DENSITY(ind) {}
	
    __host__ __device__
        bool operator()(const unsigned int thread_id) const
        {
            thrust::default_random_engine rng;
            thrust::uniform_real_distribution<float> dist(0, 1);
            rng.discard(thread_id);

            return dist(rng) <= IN_DENSITY ? true : false;
        }
};

void visualize_input_generated_on_device(thrust::device_vector<bool>& in_vector, UInt* pot_pools_host, const UInt MAX_CONNECTED, const UInt SP_SIZE)
{
	printf("INPUT\n");
	thrust::copy(in_vector.begin(), in_vector.end(), std::ostream_iterator<bool>(std::cout, " "));
	printf("\n");
	// This overlows stdout buffer (better write to a file if necessary)
	// printf("POTENTIAL POOLS");
	// for(int i=0; i<SP_SIZE; i++)
	// {
	// 	for(int j=0; j<MAX_CONNECTED; j++)
	// 		printf("%d \t", pot_pools_host[i*MAX_CONNECTED+j]);
	// 	printf("\n");
	// }


}

void visualize_output(bool* cols_host, const UInt SP_SIZE, UInt BLOCK_SIZE)
{
	printf("OUTPUT\n");
	for(int i=0; i<SP_SIZE; i++)
	{
		printf("%d ", cols_host[i]);
		if(i % BLOCK_SIZE == 0 && i > 0)
			printf("\n");
	}
	printf("\n");
	
	// The final sparsity will approach target with increasing block size
	int ones = 0;
	for(int i=0; i < SP_SIZE; i++)
		if(cols_host[i] > 0) ones++;
	printf("Sparsity: %f \n", (Real)ones/SP_SIZE);

}

int main(int argc, const char * argv[])
{
	srand(time(NULL));
	
    // construct input args
    args ar;
	ar.iteration_num=0;
	ar.learn=true;
	ar.localAreaDensity=0.02; // SP density after inhibition
    ar.potentialPct=0.1; // 
    ar.connectedPct=0.5;
    ar.stimulusThreshold=0;
    ar.synPermTrimThreshold=0.025;
    ar.synPermMax=1.0;
    ar.synPermConnected=0.1;
	ar.synPermActiveInc=0.05;
	ar.synPermInactiveDec=0.008;
	ar.synPermBelowStimulusInc=ar.synPermConnected / 10.0;
	ar.dutyCyclePeriod=1000;
	ar.boostStrength=0.05; // 0 means no boosting
	ar.minPctOdc=0.001;
	ar.update_period=50;
	
	ar.SP_SIZE = 32768;
	ar.IN_SIZE = 131072;
	ar.BLOCK_SIZE = 1024;
	ar.NUM_BLOCKS = ar.SP_SIZE/ar.BLOCK_SIZE;
	ar.IN_BLOCK_SIZE = ar.IN_SIZE/ar.NUM_BLOCKS; // Size of chunk of input processed by a single cuda block
	ar.MAX_CONNECTED = 1024;
	ar.IN_DENSITY = 0.5; // Density of input connections
	ar.num_connected = std::floor(ar.MAX_CONNECTED*ar.connectedPct);

	// Host memory allocation
    bool* cols_host = (bool*) malloc(ar.SP_SIZE*sizeof(bool));
	UInt* pot_pools_host = (UInt*) malloc(ar.SP_SIZE*ar.num_connected*sizeof(UInt));
 	pot_pools_host = generatePotentialPoolsUsingShuffle(pot_pools_host, ar.SP_SIZE, ar.IN_BLOCK_SIZE, ar.num_connected);

	// Global memory pointers
	args* ar_dev;

	// Global memory allocation
    checkError( hipMalloc((void **) &ar_dev, sizeof(ar)) );

	size_t pot_dev_pitch_in_bytes, per_dev_pitch_in_bytes;
	checkError( hipMallocPitch((void **) &ar.pot_dev, &pot_dev_pitch_in_bytes, ar.num_connected*sizeof(UInt), ar.SP_SIZE) );
	checkError( hipMallocPitch((void **) &ar.per_dev, &per_dev_pitch_in_bytes, ar.num_connected*sizeof(Real), ar.SP_SIZE) );
	ar.pot_dev_pitch = pot_dev_pitch_in_bytes / sizeof(UInt);
	ar.per_dev_pitch = per_dev_pitch_in_bytes / sizeof(Real);

	checkError( hipMalloc((void **) &ar.boosts_dev, ar.SP_SIZE*ar.num_connected*sizeof(Real)) );
    checkError( hipMalloc((void **) &ar.olaps_dev, ar.SP_SIZE*sizeof(UInt)) );
    checkError( hipMalloc((void **) &ar.cols_dev, ar.SP_SIZE*sizeof(bool)) );
	checkError( hipMalloc((void **) &ar.numPot_dev, ar.SP_SIZE*sizeof(UInt)) );
    checkError( hipMalloc((void **) &ar.odc_dev, ar.MAX_CONNECTED*ar.SP_SIZE*sizeof(Real)) );
    checkError( hipMalloc((void **) &ar.adc_dev, ar.MAX_CONNECTED*ar.SP_SIZE*sizeof(Real)) );
	checkError( hipMalloc((void **) &ar.minOdc_dev, ar.NUM_BLOCKS*sizeof(Real)) );
	checkError( hipMalloc((void **) &ar.dev_states, ar.SP_SIZE*ar.BLOCK_SIZE*sizeof(hiprandState)) );

	// Global memory initialization

	setup_kernel<<<ar.NUM_BLOCKS, ar.BLOCK_SIZE>>>(ar.dev_states);

	// Permanences
	generatePermanences<<<ar.SP_SIZE, ar.num_connected>>>(ar.per_dev, ar.per_dev_pitch, ar.connectedPct, ar.synPermConnected, ar.synPermMax, ar.dev_states);

	// Boosts
	thrust::device_ptr<float> boosts_ptr(ar.boosts_dev);
	thrust::fill(boosts_ptr, boosts_ptr+ar.SP_SIZE*ar.num_connected*sizeof(Real), 1.0);
	
	// Number of potentialy connected synapses - unnecessary if we want it variable
	thrust::device_ptr<UInt> num_ptr(ar.numPot_dev);
	thrust::fill(num_ptr, num_ptr+ar.SP_SIZE*sizeof(UInt), ar.num_connected);

	// Input
	thrust::device_vector<bool> in_vector(ar.IN_SIZE);

	thrust::counting_iterator<unsigned int> index_sequence_begin(0);

    thrust::transform(index_sequence_begin,
            index_sequence_begin + ar.IN_SIZE,
            in_vector.begin(),
            prg(ar.IN_DENSITY));

	ar.in_dev = thrust::raw_pointer_cast(&in_vector[0]);

	visualize_input_generated_on_device(in_vector, pot_pools_host, ar.num_connected, ar.SP_SIZE);

	// Memcpy to device
    checkError( hipMemcpy(ar_dev, (void**) &ar, sizeof(ar), hipMemcpyHostToDevice) );
    checkError( hipMemcpy2D(ar.pot_dev, pot_dev_pitch_in_bytes, pot_pools_host, ar.num_connected*sizeof(UInt), ar.num_connected*sizeof(UInt), ar.SP_SIZE, hipMemcpyHostToDevice) );

	// Kernel call
	size_t sm = ar.BLOCK_SIZE*(2*sizeof(Real) + sizeof(UInt)) + ar.IN_BLOCK_SIZE*sizeof(bool);
    compute<<<ar.NUM_BLOCKS, ar.BLOCK_SIZE, sm>>>(ar_dev);
    
	// Memcpy from device
    checkError( hipMemcpy(cols_host, ar.cols_dev, ar.SP_SIZE*sizeof(bool), hipMemcpyDeviceToHost)); 

	visualize_output(cols_host, ar.SP_SIZE, ar.BLOCK_SIZE);

	hipFree(ar.cols_dev); 
	hipFree(ar.pot_dev);
   	hipFree(ar.per_dev); 
	hipFree(ar.boosts_dev);
	hipFree(ar.odc_dev); 
	hipFree(ar.adc_dev); 
	hipFree(ar.numPot_dev);
    
	return 0;
}
