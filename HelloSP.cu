#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <algorithm>
#include <ctime>
#include <cstdlib>
#include <cmath>
#include <random>
#include <assert.h>

#include "SpatialPooler.cu"

using namespace std;

typedef unsigned int UInt;
typedef float Real;

UInt* generatePotentialPools(int cols, const UInt IN_BLOCK_SIZE, Real potentialPct, const UInt MAX_CONNECTED, UInt* numPotential)
{
    UInt* potentialPools = new UInt[cols*MAX_CONNECTED];
    int connected = 0;
    for(int i=0; i < cols; i++)
    {
    	connected = 0;
		// Generated indeces should be in (0,IN_BLOCK_SIZE) and their count should be <= MAX_CONNECTED and around potentialPct*IN_BLOCK_SIZE
        for(int j=0; j < IN_BLOCK_SIZE; j++)
        {
            if((Real)(rand()%100)/100 <= potentialPct && connected < MAX_CONNECTED)
            {
                potentialPools[i*MAX_CONNECTED + connected++] = j; 
            }
        }
		numPotential[i] = connected;
    }
    return potentialPools;
}

Real initPermanencesConnected(Real synPermConnected_, Real synPermMax_)
{
	Real p = synPermConnected_ +
	             (synPermMax_ - synPermConnected_)*((Real)((rand()%100))/100);
	return p;
}

Real initPermanencesNotConnected(Real synPermConnected_)
{
	Real p = synPermConnected_ * (Real)((rand()%100))/100;
	return p;
}

Real* generatePermanences(int cols, int inputSize, UInt* potential, Real connectedPct,
		Real synPermConnected_, Real synPermMax_, const UInt MAX_CONNECTED, UInt* numPotential,
	   	const UInt BLOCK_SIZE, const UInt IN_BLOCK_SIZE)
{
    Real* permanences = new Real[cols*MAX_CONNECTED];
	int connected = 0;
	int curr_block = 0;
    bool found = false;

	for(int i=0; i < cols; i++)
	{
		connected = 0;
		// We need to only go through the input block corresponding to the current column
		// This means we need to convert current column to the input block number
		curr_block = floor(i / BLOCK_SIZE);
		// j is the global index of connection in the input matrix
		for(int j=curr_block*IN_BLOCK_SIZE; j < curr_block*IN_BLOCK_SIZE + IN_BLOCK_SIZE; j++)
		{
			// Find if this input is potentially connected with this column
			found=false;
            for(int k=0; k < numPotential[i]; k++)
            {
                if(potential[i*MAX_CONNECTED+k] == j % IN_BLOCK_SIZE) {
					found = true;
					break;
				}
            }
			// If there is, decide if it will be. The structure of the data is as follows:
		    // potential[col][index of the synapse on the segment] = index of input in the block
			// permanences[col][index of the synapse on the segment] = permanence of the synapse
            if(found)
            {
                if((Real)(rand()%100)/100 <= connectedPct)
                {
                    permanences[i*MAX_CONNECTED+connected++] = initPermanencesConnected(synPermConnected_, synPermMax_);
                }
                else
                {
                    permanences[i*MAX_CONNECTED+connected++] = initPermanencesNotConnected(synPermConnected_);
                }
            }
		}
	}
	return permanences;
}

// TO BE DELETED
// There should also be a parameter to raise permanences so that minimum number of synapses is connected.
UInt** computeConnected(Real** permanences, UInt** potential, UInt cols, UInt inputSize,
		Real synPermConnected_, const UInt MAX_CONNECTED, UInt* numPotential)
{
	UInt** connected_arr = new UInt*[cols];
	int connected = 0;
	for(int i=0; i < inputSize; i++)
	{
		connected = 0;
        connected_arr[i] = new UInt[MAX_CONNECTED];
		for(int j=0; j < numPotential[i]; j++)
		{
			if(permanences[i][j] < synPermConnected_)
			{
				connected_arr[i][connected++] = j;
			}
		}
	}
	return connected_arr;
}

void generate01(bool* ar, size_t size, Real inDensity)
{
	for(int i=0; i < size; i++)
	{
		ar[i] = (Real)(rand()%100)/100 <= inDensity ? 1 : 0;
	}
}

void visualize_input(bool* in_host, UInt* potentialPools, Real* permanences, UInt* numPotential, const UInt IN_SIZE, const UInt SP_SIZE, const UInt IN_BLOCK_SIZE, const UInt MAX_CONNECTED)
{
	printf("POTENTIAL CONNECTIONS WITH PERMANENCES\n");
	for(int i=0; i<SP_SIZE; i++)
	{
		for(int j=0; j<MAX_CONNECTED; j++)
			printf("%d \t", potentialPools[i*MAX_CONNECTED+j]);
		printf("\n");
		for(int j=0; j<numPotential[i]; j++)
			printf("%.2f\t", permanences[i*MAX_CONNECTED+j]);
		printf("\n");
		printf("%d \n", numPotential[i]);
	}

	printf("INPUT SDR\n");
	for(int i=0; i<IN_SIZE; i++)
	{
		printf("%d ", in_host[i]);
		if(i % IN_BLOCK_SIZE == 0 && i > 0)
			printf("\n");
	}
	printf("\n");
}

void visualize_output(bool* cols_host, const UInt SP_SIZE)
{
	// The final sparsity will approach target with increasing block size
	int ones = 0;
	for(int i=0; i < SP_SIZE; i++)
		if(cols_host[i] > 0) ones++;
	printf("Sparsity: %f \n", (Real)ones/SP_SIZE);
}

void printErrorMessage(hipError_t error, int memorySize){
    printf("==================================================\n");
    printf("MEMORY ERROR  : %s\n", hipGetErrorString(error));
    printf("==================================================\n");
}

int main(int argc, const char * argv[])
{
	const UInt SP_SIZE = 524288;
	const UInt IN_SIZE = 1048576;
	const UInt BLOCK_SIZE = 64; // Two warps
	const UInt NUM_BLOCKS = SP_SIZE/BLOCK_SIZE;
	const UInt IN_BLOCK_SIZE = IN_SIZE/NUM_BLOCKS; // Size of chunk of input processed by a single cuda block
	const UInt MAX_CONNECTED = 16;
    const Real IN_DENSITY = 0.5; // Density of input connections
    srand(time(NULL));

	size_t sm = BLOCK_SIZE*(2*sizeof(Real) + sizeof(UInt)) + IN_BLOCK_SIZE*sizeof(bool);

    // construct input args
    args ar;
	ar.iteration_num=0;
	ar.learn=true;
	ar.localAreaDensity=0.02; // SP density after inhibition
    ar.potentialPct=0.5; // 
    ar.connectedPct=0.5;
    ar.stimulusThreshold=0;
    ar.synPermTrimThreshold=0.025;
    ar.synPermMax=1.0;
    ar.synPermConnected=0.1;
	ar.synPermActiveInc=0.05;
	ar.synPermInactiveDec=0.008;
	ar.synPermBelowStimulusInc=ar.synPermConnected / 10.0;
	ar.dutyCyclePeriod=1000;
	ar.boostStrength=0.05; // 0 means no boosting
	ar.minPctOdc=0.001;
	ar.update_period=50;
	ar.SP_SIZE = SP_SIZE;
	ar.MAX_CONNECTED = MAX_CONNECTED;
	ar.IN_BLOCK_SIZE = IN_BLOCK_SIZE;

	// Host memory pointers
    bool* cols_host = new bool[SP_SIZE];
	bool* in_host = new bool[IN_SIZE];
    UInt* potentialPools;
	Real* permanences;
	Real* boosts = new Real[SP_SIZE*MAX_CONNECTED];
	UInt* numPotential = new UInt[SP_SIZE];
	UInt* numConnected = new UInt[SP_SIZE];

	// Host memory allocation	
	std::fill_n(boosts, SP_SIZE*MAX_CONNECTED, 1);
	std::fill_n(numPotential, SP_SIZE, 0);
	std::fill_n(numConnected, SP_SIZE, 0);

	potentialPools = generatePotentialPools(SP_SIZE, IN_BLOCK_SIZE, ar.potentialPct, MAX_CONNECTED, numPotential);
	permanences = generatePermanences(SP_SIZE, IN_SIZE, potentialPools, ar.connectedPct, ar.synPermConnected, ar.synPermMax, MAX_CONNECTED, numPotential,
					BLOCK_SIZE, IN_BLOCK_SIZE);
	generate01(in_host, IN_SIZE, IN_DENSITY);

	// visualize_input(in_host, potentialPools, permanences, numPotential, IN_SIZE, SP_SIZE, IN_BLOCK_SIZE, MAX_CONNECTED);

	// Global memory pointers
	args* ar_dev;

	// Global memory allocation
	hipError_t result;
    result = hipMalloc((void **) &ar_dev, sizeof(ar)); if(result) printErrorMessage(result, 0);
    result = hipMalloc((void **) &ar.in_dev, IN_SIZE*sizeof(bool)); if(result) printErrorMessage(result, 0);
    result = hipMalloc((void **) &ar.olaps_dev, SP_SIZE*sizeof(UInt)); if(result) printErrorMessage(result, 0);
    result = hipMalloc((void **) &ar.cols_dev, SP_SIZE*sizeof(bool)); if(result) printErrorMessage(result, 0);
	result = hipMalloc((void **) &ar.numPot_dev, SP_SIZE*sizeof(UInt)); if(result) printErrorMessage(result, 0);
    result = hipMalloc((void **) &ar.pot_dev, MAX_CONNECTED*SP_SIZE*sizeof(UInt)); if(result) printErrorMessage(result, 0); // width, height, x, y 
    result = hipMalloc((void **) &ar.per_dev, MAX_CONNECTED*SP_SIZE*sizeof(Real)); if(result) printErrorMessage(result, 0); 
    result = hipMalloc((void **) &ar.odc_dev, MAX_CONNECTED*SP_SIZE*sizeof(Real)); if(result) printErrorMessage(result, 0); 
    result = hipMalloc((void **) &ar.adc_dev, MAX_CONNECTED*SP_SIZE*sizeof(Real)); if(result) printErrorMessage(result, 0); 
    result = hipMalloc((void **) &ar.boosts_dev, MAX_CONNECTED*SP_SIZE*sizeof(Real)); if(result) printErrorMessage(result, 0); 
    result = hipMalloc((void **) &ar.minOdc_dev, NUM_BLOCKS*sizeof(Real)); if(result) printErrorMessage(result, 0); 

	// Memcpy to device
    result = hipMemcpy(ar_dev, &ar, sizeof(ar), hipMemcpyHostToDevice); if(result) printErrorMessage(result, 0);
    result = hipMemcpy(ar.in_dev, in_host, IN_SIZE*sizeof(bool), hipMemcpyHostToDevice); if(result) printErrorMessage(result, 0);
    result = hipMemcpy(ar.numPot_dev, numPotential, SP_SIZE*sizeof(UInt), hipMemcpyHostToDevice); if(result) printErrorMessage(result, 0);
    result = hipMemcpy(ar.pot_dev, potentialPools, MAX_CONNECTED*SP_SIZE*sizeof(UInt), hipMemcpyHostToDevice); if(result) printErrorMessage(result, 0);
    result = hipMemcpy(ar.per_dev, permanences, MAX_CONNECTED*SP_SIZE*sizeof(Real), hipMemcpyHostToDevice); if(result) printErrorMessage(result, 0);
    result = hipMemcpy(ar.boosts_dev, boosts, MAX_CONNECTED*SP_SIZE*sizeof(Real), hipMemcpyHostToDevice); if(result) printErrorMessage(result, 0);

	// Kernel call
    compute<<<NUM_BLOCKS, BLOCK_SIZE, sm>>>(ar_dev);

    // Memcpy from device
    result = hipMemcpy(cols_host, ar.cols_dev, SP_SIZE*sizeof(bool), hipMemcpyDeviceToHost); if(result) printErrorMessage(result, 0); 

	visualize_output(cols_host, SP_SIZE);

    hipFree(ar.in_dev); hipFree(ar.cols_dev); hipFree(ar.pot_dev); hipFree(ar.per_dev); hipFree(ar.boosts_dev);
	hipFree(ar.odc_dev); hipFree(ar.adc_dev); hipFree(ar.numPot_dev);

    return 0;
}
