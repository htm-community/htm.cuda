#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

using namespace std;

typedef unsigned int UInt;
typedef float Real;

struct args
{
	// Parameters
    UInt inputPitch;
    UInt stimulusThreshold;
    Real potentialPct;
    Real connectedPct;
	Real localAreaDensity;
    Real synPermTrimThreshold;
    Real synPermMax;
    Real synPermConnected;
	Real synPermActiveInc;
	Real synPermInactiveDec;
	Real synPermBelowStimulusInc;
	UInt dutyCyclePeriod;
	Real boostStrength;
	Real minPctOdc;
	bool learn;

	// Data
	bool* in_dev;
    bool* cols_dev;
	UInt* olaps_dev;
	UInt* pot_dev;
	Real* per_dev;
	Real* boosts_dev;
	Real* odc_dev; // odc serve to maintain same act. freq. for each col. (per block)
	Real* adc_dev; // adc serve to compute boost factors
	UInt* numPot_dev;
	Real* minOdc_dev;

	// Constants
	UInt SP_SIZE;
	UInt IN_SIZE;
	UInt BLOCK_SIZE;
	UInt NUM_BLOCKS;
	UInt IN_BLOCK_SIZE; // Size of chunk of input processed by a single cuda block
	UInt MAX_CONNECTED;
	Real IN_DENSITY; // Density of input connections
	UInt num_connected;

	// Array pitches
	size_t pot_dev_pitch;
	size_t per_dev_pitch;

	// Bookkeeping vars
	UInt iteration_num;
	UInt update_period;

	hiprandState* dev_states;
};

__global__ void setup_kernel(hiprandState *state)
{
    int id = threadIdx.x + blockIdx.x*blockDim.x;
    hiprand_init(727612, id, 0, &state[id]);
}

__device__
inline void random_swap(volatile UInt& a, volatile UInt& b, hiprandState& state)
{
	// if(curand(state) & 1)
	if(hiprand_uniform(&state) < 0.5)
	{
		UInt temp;
		temp = a;
		a = b;
		b = temp;
	}
}

__global__
void generatePotentialPools(UInt* pot_dev, size_t pot_dev_pitch, UInt num_connected, UInt* input_indeces, hiprandState* states, UInt IN_BLOCK_SIZE)
{
	UInt tx = threadIdx.x;
	UInt BLOCK_SIZE = blockDim.x;
	hiprandState localState = states[threadIdx.x + blockIdx.x*blockDim.x];
	extern __shared__ volatile UInt shared[];

	shared[tx] = input_indeces[tx];

    int id = BLOCK_SIZE;	
	float x = 0;
	while(id < IN_BLOCK_SIZE - tx)
	{
		// x = (float) (curand(&localState) % 100) / 100;
		x = hiprand_uniform(&localState);
		// if(x > (float) BLOCK_SIZE / IN_BLOCK_SIZE)
		if(x < 0.5)
		{
			shared[tx] = input_indeces[tx+id];
		}
		id += BLOCK_SIZE;
	}

	__syncthreads();

	// Do reduction on shared
	if(BLOCK_SIZE >= 1024)
	{ 
		if(tx < 512) 
		{ 
			random_swap(shared[tx], shared[tx+512], localState); 
		} 
		__syncthreads(); 
	}

	if(BLOCK_SIZE >= 512)
	{ 
		if(tx < 256) 
		{ 
			random_swap(shared[tx], shared[tx+256], localState); 
		} 
		__syncthreads(); 
	}
    if(BLOCK_SIZE >= 256)
   	{ 
		if(tx < 128) 
		{ 
			random_swap(shared[tx], shared[tx+128], localState); 
		} 
		__syncthreads(); 
	}
    if(BLOCK_SIZE >= 128)
   	{ 
		if(tx < 64) 
		{ 
			random_swap(shared[tx], shared[tx+64], localState); 
		} 
		__syncthreads(); 
	}

	if(tx < 32) 
    {
        if(BLOCK_SIZE >= 64) 
			random_swap(shared[tx], shared[tx+32], localState);
        if(BLOCK_SIZE >= 32) 
			random_swap(shared[tx], shared[tx+16], localState);
        if(BLOCK_SIZE >= 16) 
			random_swap(shared[tx], shared[tx+8], localState);
        if(BLOCK_SIZE >= 8) 
			random_swap(shared[tx], shared[tx+4], localState);
        if(BLOCK_SIZE >= 4)
			random_swap(shared[tx], shared[tx+2], localState);
        if(BLOCK_SIZE >= 2) 
			random_swap(shared[tx], shared[tx+1], localState);
    }

	__syncthreads();
	
	if(tx < num_connected)
		pot_dev[blockIdx.x*pot_dev_pitch + tx] = shared[tx];
}

__global__
void generatePermanences(Real* per_dev, size_t per_dev_pitch, Real connectedPct, Real synPermConnected, Real synPermMax, hiprandState* states)
{
	UInt col = blockIdx.x;
	UInt tx = threadIdx.x;
	hiprandState localState = states[col*blockDim.x + tx];
	bool connected = (Real) hiprand_uniform(&localState) <= connectedPct;
	per_dev[col*per_dev_pitch + tx] = connected ? synPermConnected + (synPermMax - synPermConnected)*((Real) hiprand_uniform(&localState)) :
													synPermConnected * (Real)hiprand_uniform(&localState);
}

__device__
void calculateOverlap(volatile UInt* olaps_sh, volatile bool* in_sh, bool* in_dev, UInt* pot_dev, size_t pot_dev_pitch, Real* per_dev, size_t per_dev_pitch, Real* boosts_dev, Real threshold, UInt numConnected, const UInt IN_BLOCK_SIZE)
{
	UInt tx = threadIdx.x;
   	UInt sp_idx = blockDim.x*blockIdx.x + tx; // Global index in the SP
	UInt in_block_start = IN_BLOCK_SIZE*blockIdx.x;
	olaps_sh[tx] = 0;

	for(int i = 0; i < IN_BLOCK_SIZE - tx; i += blockDim.x)
		in_sh[tx + i] = in_dev[in_block_start + tx + i]; 

	__syncthreads();

    for(int i=0; i < numConnected; i++)
    {
		UInt bl_idx = pot_dev[sp_idx*pot_dev_pitch+i]; // Index of block-specific input
		if(in_sh[bl_idx] && (per_dev[sp_idx*per_dev_pitch + i] > threshold))
        	olaps_sh[tx] += boosts_dev[sp_idx+i];
    }

// 	__syncthreads();
// 
// 	olaps_sh[tx] = olaps;
}

__device__
void inhibitColumns(volatile UInt* olaps_sh, bool* cols_dev, volatile Real* active_sh, bool &active, Real sparsity)
{
    int tx = threadIdx.x;
	int numLarger = 0;
	active = false;
	
	for(int i=0; i < blockDim.x; i++)
	{
		if(olaps_sh[i] > olaps_sh[tx]) numLarger++;
	}
	if(numLarger < sparsity * (Real) blockDim.x && numLarger > 0) active = true;

	__syncthreads();

	cols_dev[blockIdx.x*blockDim.x + tx] = active;
	active_sh[tx] = active;
}

__device__
void adaptSynapses(bool* in_dev, UInt* pot_dev, Real* per_dev, Real synPermActiveInc, Real synPermInactiveDec, bool active, const UInt inBlockSize, UInt num_connected, const size_t per_dev_pitch, const size_t pot_dev_pitch)
{
    int tx = threadIdx.x;
   	int sp_idx = blockDim.x*blockIdx.x + tx;
	if(active)
	{
		for(int i=0; i < num_connected; i++)
    	{
			int in_idx = pot_dev[sp_idx*pot_dev_pitch+i];
			if(in_dev[inBlockSize*blockIdx.x + in_idx])
				per_dev[sp_idx*per_dev_pitch+i] = min(1.0, per_dev[sp_idx*per_dev_pitch+i]+synPermActiveInc);
			else
				per_dev[sp_idx*per_dev_pitch+i] = max(per_dev[sp_idx*per_dev_pitch+i]-synPermInactiveDec, 0.0);
    	}
	}
}

__device__
void updateDutyCycles(Real* odc_dev, Real* adc_dev, volatile UInt* olaps_sh, bool active, UInt iteration_num, UInt dutyCyclePeriod)
{
    int tx = threadIdx.x;

	// Let grow divisor only to a dutyCyclePeriod to not make the update increasingly negligible
	Real period = dutyCyclePeriod > iteration_num ? iteration_num : dutyCyclePeriod;

	odc_dev[blockDim.x*blockIdx.x+tx] = (odc_dev[blockDim.x*blockIdx.x+tx]*(period-1) + (Real)(olaps_sh[tx] > 0)) / period;
	adc_dev[blockDim.x*blockIdx.x+tx] = (odc_dev[blockDim.x*blockIdx.x+tx]*(period-1) + (Real)active) / period;
}

__device__
void averageActivity(volatile Real* active_sh)
{
	Real avg = 0;
	for(int i=0; i < blockDim.x; i++)
	{
		avg += active_sh[i];
	}
	active_sh[threadIdx.x] = avg / (Real)blockDim.x;
}

__device__
void averageActivityReduction(volatile Real* active_sh)
{
	int tx = threadIdx.x;
	UInt BLOCK_SIZE = blockDim.x;

	if(BLOCK_SIZE >= 1024)
	{ 
		if(tx < 512) 
		{ 
			active_sh[tx] += active_sh[tx+512]; 
		} 
		__syncthreads(); 
	}
	if(BLOCK_SIZE >= 512)
	{ 
		if(tx < 256) 
		{ 
			active_sh[tx] += active_sh[tx+256]; 
		} 
		__syncthreads(); 
	}
    if(BLOCK_SIZE >= 256)
   	{ 
		if(tx < 128) 
		{ 
			active_sh[tx] += active_sh[tx+128]; 
		} 
		__syncthreads(); 
	}
    if(BLOCK_SIZE >= 128)
   	{ 
		if(tx < 64) 
		{ 
			active_sh[tx] += active_sh[tx+64]; 
		} 
		__syncthreads(); 
	}

    if(tx < 32) 
    {
        if(BLOCK_SIZE >= 64) 
			active_sh[tx] += active_sh[tx+32];
        if(BLOCK_SIZE >= 32) 
			active_sh[tx] += active_sh[tx+16];
        if(BLOCK_SIZE >= 16) 
			active_sh[tx] += active_sh[tx+8];
        if(BLOCK_SIZE >= 8) 
			active_sh[tx] += active_sh[tx+4];
        if(BLOCK_SIZE >= 4)
			active_sh[tx] += active_sh[tx+2];
        if(BLOCK_SIZE >= 2) 
			active_sh[tx] += active_sh[tx+1];
    }

	__syncthreads();

	// According to https://devblogs.nvidia.com/using-shared-memory-cuda-cc/, this should result in a broadcast
    active_sh[tx] = active_sh[0] / BLOCK_SIZE;
}

__device__
void updateBoosts(Real* adc_dev, Real* boosts_dev, Real targetDensity, Real boostStrength)
{
    int sp_idx = blockIdx.x*blockDim.x+threadIdx.x;
	boosts_dev[sp_idx] = exp((targetDensity - adc_dev[sp_idx])*boostStrength);
}

__device__
void bumpUpColumnsWithWeakOdc(Real* odc_dev, Real* per_dev, UInt* numPot, Real* minOdc_dev, Real synPermBelowStimulusInc, const UInt MAX_CONNECTED)
{
	int tx = threadIdx.x;
    int sp_idx = blockIdx.x*blockDim.x+tx;

	if(odc_dev[sp_idx] < minOdc_dev[blockIdx.x]) {
		for(int i=0; i<numPot[sp_idx]; i++)
			per_dev[tx*MAX_CONNECTED+i] += synPermBelowStimulusInc;
	}
}

__device__
void updateMinOdc(Real* odc_dev, volatile Real* odc_sh, Real* minOdc_dev, Real minPctOdc, const UInt SP_SIZE)
{
	Real maxOdc = 0;
	for(int i=0; i<SP_SIZE; i++)
		maxOdc = odc_dev[i] > maxOdc ? odc_dev[i] : maxOdc;
	if(threadIdx.x == 0)
		minOdc_dev[blockIdx.x] = minPctOdc * maxOdc;
}

__device__
void updateMinOdcReduction(Real* odc_dev, volatile Real* odc_sh, Real* minOdc_dev, Real minPctOdc, const UInt SP_SIZE)
{
	int tx = threadIdx.x;
	int sp_idx = blockDim.x*blockIdx.x + threadIdx.x;
	UInt BLOCK_SIZE = blockDim.x;

	odc_sh[tx] = odc_dev[sp_idx];

	if(BLOCK_SIZE >= 1024)
	{ 
		if(tx < 512) 
		{ 
			odc_sh[tx] = max(odc_sh[tx], odc_sh[tx+512]); 
		} 
		__syncthreads(); 
	}
	if(BLOCK_SIZE >= 512)
	{ 
		if(tx < 256) 
		{ 
			odc_sh[tx] = max(odc_sh[tx], odc_sh[tx+256]); 
		} 
		__syncthreads(); 
	}
    if(BLOCK_SIZE >= 256)
   	{ 
		if(tx < 128) 
		{ 
			odc_sh[tx] = max(odc_sh[tx], odc_sh[tx+128]); 
		} 
		__syncthreads(); 
	}
    if(BLOCK_SIZE >= 128)
   	{ 
		if(tx < 64) 
		{ 
			odc_sh[tx] = max(odc_sh[tx], odc_sh[tx+64]); 
		} 
		__syncthreads(); 
	}

    if(tx < 32) 
    {
        if(BLOCK_SIZE >= 64) 
			odc_sh[tx] = max(odc_sh[tx], odc_sh[tx+32]);
        if(BLOCK_SIZE >= 32) 
			odc_sh[tx] = max(odc_sh[tx], odc_sh[tx+16]);
        if(BLOCK_SIZE >= 16) 
			odc_sh[tx] = max(odc_sh[tx], odc_sh[tx+8]);
        if(BLOCK_SIZE >= 8) 
			odc_sh[tx] = max(odc_sh[tx], odc_sh[tx+4]);
        if(BLOCK_SIZE >= 4)
			odc_sh[tx] = max(odc_sh[tx], odc_sh[tx+2]);
        if(BLOCK_SIZE >= 2) 
			odc_sh[tx] = max(odc_sh[tx], odc_sh[tx+1]);
    }

	if(threadIdx.x == 0)
		minOdc_dev[blockIdx.x] = minPctOdc * odc_sh[0];
}


__global__
void compute(args* ar_ptr)
{
	// Global memory pointers
    // bool* cols_dev = (bool*) data;
	// bool* in_dev = &cols_dev[SP_SIZE];
	// UInt* pot_dev = (UInt*) &in_dev[IN_SIZE];
	// UInt* numPot_dev = &pot_dev[SP_SIZE*MAX_CONNECTED];
	// Real* per_dev = (Real*) &numPot_dev[SP_SIZE];
	// Real* boosts_dev = &per_dev[SP_SIZE*MAX_CONNECTED];
	// UInt* olaps_dev = (UInt*) &boosts_dev[SP_SIZE*MAX_CONNECTED];
	// Real* odc_dev = (Real*) &olaps_dev[SP_SIZE]; // odc serve to maintain same act. freq. for each col. (per block)
	// Real* adc_dev =  &odc_dev[MAX_CONNECTED*SP_SIZE]; // adc serve to compute boost factors
	// Real* minOdc_dev = &adc_dev[MAX_CONNECTED*SP_SIZE]; // Stores minumum overlap duty cycles per block 

	
	if (blockIdx.x == 0 && threadIdx.x == 0) 
		ar_ptr->iteration_num++;
	
	args ar = *ar_ptr;

	bool active = false;
	Real avg_act = 0;

    extern __shared__ volatile UInt shared[];
	volatile UInt* olaps_sh = &shared[0];
	volatile Real* active_sh = (Real*)&shared[blockDim.x];
	volatile Real* odc_sh = &active_sh[blockDim.x];
	volatile bool* in_sh = (bool*) &odc_sh[blockDim.x];

	// calculateOverlap(ar.in_dev, in_sh, ar.pot_dev, ar.per_dev, ar.boosts_dev, ar.numPot_dev, olaps_sh, ar.synPermConnected, ar.IN_BLOCK_SIZE, ar.MAX_CONNECTED);

    calculateOverlap(olaps_sh, in_sh, ar.in_dev, ar.pot_dev, ar.pot_dev_pitch, ar.per_dev, ar.per_dev_pitch, ar.boosts_dev, ar.synPermConnected, ar.num_connected, ar.IN_BLOCK_SIZE);
	
	__syncthreads();

	inhibitColumns(olaps_sh, ar.cols_dev, active_sh, active, ar.localAreaDensity);
	
	__syncthreads();

	adaptSynapses(ar.cols_dev, ar.pot_dev, ar.per_dev, ar.synPermActiveInc, ar.synPermInactiveDec, active, ar.IN_BLOCK_SIZE, ar.num_connected, ar.per_dev_pitch, ar.pot_dev_pitch);

	updateDutyCycles(ar.odc_dev, ar.adc_dev, olaps_sh, active, ar.iteration_num, ar.dutyCyclePeriod);

	averageActivityReduction(active_sh);

	__syncthreads();

	updateBoosts(ar.adc_dev, ar.boosts_dev, avg_act, ar.boostStrength);

	bumpUpColumnsWithWeakOdc(ar.odc_dev, ar.per_dev, ar.numPot_dev, ar.minOdc_dev, ar.synPermBelowStimulusInc, ar.MAX_CONNECTED);

	if(ar.iteration_num % ar.update_period == 0)
		updateMinOdc(ar.odc_dev, ar.odc_dev, ar.minOdc_dev, ar.minPctOdc, ar.SP_SIZE);
}

__global__
void calculateOverlap_wrapper(bool* in_dev, UInt* pot_dev, Real* per_dev, Real* boosts_dev, UInt* numPot_dev, Real threshold, const UInt inBlockSize, const UInt MAX_CONNECTED, UInt* olaps_dev, const UInt SP_SIZE, size_t pot_dev_pitch, size_t per_dev_pitch)
{
	extern __shared__ volatile UInt shared[];
	volatile UInt* olaps_sh = &shared[0];
	volatile bool* in_sh = (bool*) &olaps_sh[blockDim.x];

	calculateOverlap(olaps_sh, in_sh, in_dev, pot_dev, pot_dev_pitch, per_dev, per_dev_pitch, boosts_dev, threshold, MAX_CONNECTED, inBlockSize);

	if(blockDim.x*blockIdx.x+threadIdx.x < SP_SIZE)
		olaps_dev[blockDim.x*blockIdx.x+threadIdx.x] = olaps_sh[threadIdx.x];
}

__global__
void inhibitColumns_wrapper(UInt* olaps_dev, bool* cols_dev, Real localAreaDensity, const UInt BLOCK_SIZE)
{
	extern __shared__ volatile UInt shared[];
	volatile UInt* olaps_sh = &shared[0];
	volatile Real* active_sh = (Real*) &olaps_sh[BLOCK_SIZE];

	olaps_sh[threadIdx.x] = olaps_dev[threadIdx.x];

	bool active = false;

	__syncthreads();

	inhibitColumns(olaps_sh, cols_dev, active_sh, active, localAreaDensity);
}

__global__
void adaptSynapses_wrapper(bool* in_dev, UInt* pot_dev, Real* per_dev, Real synPermActiveInc, Real synPermInactiveDec, bool* active_arr, const UInt IN_BLOCK_SIZE, const size_t per_dev_pitch, const size_t pot_dev_pitch, UInt num_connected, const UInt SP_SIZE)
{
	int sp_idx = blockIdx.x*blockDim.x + threadIdx.x;
	if(sp_idx < SP_SIZE)
	{
		bool active = active_arr[sp_idx];
		adaptSynapses(in_dev, pot_dev, per_dev, synPermActiveInc, synPermInactiveDec, active, IN_BLOCK_SIZE, num_connected, per_dev_pitch, pot_dev_pitch);
	}
}

__global__
void averageActivity_wrapper(bool* cols_dev, Real* avg_dev)
{
	int tx = threadIdx.x;

	extern __shared__ volatile UInt shared[];
	volatile Real* active_sh = (Real*) &shared[0];

	active_sh[tx] = (Real) cols_dev[tx];

	averageActivityReduction(active_sh);

	avg_dev[tx] = active_sh[tx];	
}
